#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/correlative_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void CorrelativeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
 
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
                        K_, K_, M_, (Dtype)(1. / M_),
                        bottom_data, bottom_data, (Dtype)0., top_data);
}

template <typename Dtype>
void CorrelativeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,const vector<bool> &propagate_down, 
    const vector<Blob<Dtype>*>& bottom) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // CHECK_EQ(top[0]->count(),48*48)
    //  <<"inequal count!";
    // Gradient with respect to bottom data
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
        M_, K_, K_,
        (Dtype)(2. / M_), bottom[0]->gpu_data(), top_diff,
        (Dtype)0., bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(CorrelativeLayer);

}  // namespace caffe
